#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <gtest/gtest.h>
#include <cudf/copying.hpp>
#include <cudf/table/table.hpp>
#include <utilities/cudf_utils.h>
#include <cudf/column/column_factories.hpp>
#include <tests/utilities/column_utilities.cuh>
#include <cudf/utilities/type_dispatcher.hpp>

void expect_columns_prop_equal(cudf::column_view lhs, cudf::column_view rhs) {
  EXPECT_EQ(lhs.type(), rhs.type());
  EXPECT_EQ(lhs.size(), rhs.size());
  EXPECT_EQ(lhs.null_count(), rhs.null_count());
  EXPECT_EQ(lhs.nullable(), rhs.nullable());
  EXPECT_EQ(lhs.has_nulls(), rhs.has_nulls());
  EXPECT_EQ(lhs.num_children(), rhs.num_children());
}

template <typename T>
struct EmptyLikeTest : GdfTest {};

using numeric_types =
    ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double>;

TYPED_TEST_CASE(EmptyLikeTest, numeric_types);

TYPED_TEST(EmptyLikeTest, NumericTests) {
    cudf::size_type size = 10;
    cudf::mask_state state = cudf::ALL_VALID;
    std::unique_ptr<cudf::column> input = make_numeric_column(cudf::data_type{cudf::exp::type_to_id<TypeParam>()}, size, state);
    std::unique_ptr<cudf::column> expected = make_numeric_column(cudf::data_type{cudf::exp::type_to_id<TypeParam>()}, 0);
    std::unique_ptr<cudf::column> got = cudf::exp::empty_like(input->view());
    expect_columns_prop_equal(*expected, *got);
}


