#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/aggregation.hpp>
#include <rolling/rolling_detail.hpp>
#include <cudf/rolling.hpp>
#include <cudf/detail/utilities/device_operators.cuh>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/copying.hpp>

#include <rmm/device_scalar.hpp>

#include <memory>

namespace cudf {
namespace experimental {

namespace detail {

/**
 * @brief Computes the rolling window function
 *
 * @tparam ColumnType  Datatype of values pointed to by the pointers
 * @tparam agg_op  A functor that defines the aggregation operation
 * @tparam is_mean Compute mean=sum/count across all valid elements in the window
 * @tparam block_size CUDA block size for the kernel
 * @tparam has_nulls true if the input column has nulls
 * @tparam WindowIterator iterator type (inferred)
 * @param input Input column device view
 * @param output Output column device view
 * @param preceding_window_begin[in] Rolling window size iterator, accumulates from
 *                in_col[i-preceding_window] to in_col[i] inclusive
 * @param following_window_begin[in] Rolling window size iterator in the forward
 *                direction, accumulates from in_col[i] to
 *                in_col[i+following_window] inclusive
 * @param min_periods[in]  Minimum number of observations in window required to
 *                have a value, otherwise 0 is stored in the valid bit mask
 */
namespace { // anonymous

    template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, int block_size, bool is_count_op, bool source_has_nulls,
             typename WindowIterator, std::enable_if_t<!std::is_same<InputType, cudf::string_view>::value>* = nullptr>
    __launch_bounds__(block_size)
    __global__
    void gpu_rolling(column_device_view input,
            mutable_column_device_view output,
            size_type * __restrict__ output_valid_count,
            WindowIterator preceding_window_begin,
            WindowIterator following_window_begin,
            size_type min_periods,
            InputType identity)
    {
        //printf("RGSL : Coming to gpu_rolling Normal\n");
        size_type i = blockIdx.x * block_size + threadIdx.x;
        size_type stride = block_size * gridDim.x;

        size_type warp_valid_count{0};

        auto active_threads = __ballot_sync(0xffffffff, i < input.size());
        while(i < input.size())
        {
            OutputType val = agg_op::template identity<OutputType>();
            // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
            // for CUDA 10.0 and below (fixed in CUDA 10.1)
            volatile cudf::size_type count = 0;

            size_type preceding_window = preceding_window_begin[i];
            size_type following_window = following_window_begin[i];

            // compute bounds
            size_type start_index = max(0, i - preceding_window);
            size_type end_index = min(input.size(), i + following_window + 1); // exclusive

            // aggregate
            // TODO: We should explore using shared memory to avoid redundant loads.
            //       This might require separating the kernel into a special version
            //       for dynamic and static sizes.
            for (size_type j = start_index; j < end_index; j++) {
                if (!source_has_nulls || input.is_valid(j)) {
                    // Element type and output type are different for COUNT
                    OutputType element = (op == aggregation::COUNT) ? InputType{0} : input.element<InputType>(j);
                    val = agg_op{}(element, val);
                    count++;
                }
            }

            // check if we have enough input samples
            bool output_is_valid = (count >= min_periods);

            // set the mask
            cudf::bitmask_type result_mask{__ballot_sync(active_threads, output_is_valid)};

            // only one thread writes the mask
            if (0 == threadIdx.x % cudf::experimental::detail::warp_size) {
                output.set_mask_word(cudf::word_index(i), result_mask);
                warp_valid_count += __popc(result_mask);
            }

            // store the output value, one per thread
            if (output_is_valid)
                cudf::detail::store_output_functor<OutputType, op == aggregation::MEAN>{}(output.element<OutputType>(i),
                        val, count);

            // process next element 
            i += stride;
            active_threads = __ballot_sync(active_threads, i < input.size());
        }

        // sum the valid counts across the whole block  
        size_type block_valid_count = 
            cudf::experimental::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

        if(threadIdx.x == 0) {
            atomicAdd(output_valid_count, block_valid_count);
        }
    }

    template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, int block_size, bool is_count_op, bool source_has_nulls,
             typename WindowIterator, std::enable_if_t<std::is_same<InputType, cudf::string_view>::value>* = nullptr>
    __launch_bounds__(block_size)
    __global__
    void gpu_rolling(column_device_view input,
            mutable_column_device_view output,
            size_type * __restrict__ output_valid_count,
            WindowIterator preceding_window_begin,
            WindowIterator following_window_begin,
            size_type min_periods,
            InputType identity)
    {
        //printf ("RGSL : Coming to Arg Max/Min \n");
        size_type i = blockIdx.x * block_size + threadIdx.x;
        size_type stride = block_size * gridDim.x;

        size_type warp_valid_count{0};

        auto active_threads = __ballot_sync(0xffffffff, i < input.size());
        while(i < input.size())
        {
            InputType val = identity;
            OutputType val_index = (op == aggregation::ARGMIN)? ARGMIN_SENTINEL : ARGMAX_SENTINEL;
            // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
            // for CUDA 10.0 and below (fixed in CUDA 10.1)
            volatile cudf::size_type count = 0;

            size_type preceding_window = preceding_window_begin[i];
            size_type following_window = following_window_begin[i];

            // compute bounds
            size_type start_index = max(0, i - preceding_window);
            size_type end_index = min(input.size(), i + following_window + 1); // exclusive

            // aggregate
            // TODO: We should explore using shared memory to avoid redundant loads.
            //       This might require separating the kernel into a special version
            //       for dynamic and static sizes.
            for (size_type j = start_index; j < end_index; j++) {
                if (!source_has_nulls || input.is_valid(j)) {
                    if (is_count_op) {
                        count++;
                    } else {
                        // Element type and output type are different for COUNT
                        InputType element = input.element<InputType>(j);
                        val = agg_op{}(element, val);
                        if (val == element) {
                            val_index = j;
                        }
                        count++;
                    }
                }
            }

            // check if we have enough input samples
            bool output_is_valid = (count >= min_periods);

            // set the mask
            cudf::bitmask_type result_mask{__ballot_sync(active_threads, (is_count_op)? output_is_valid: true)};

            // only one thread writes the mask
            if (0 == threadIdx.x % cudf::experimental::detail::warp_size) {
                output.set_mask_word(cudf::word_index(i), result_mask);
                warp_valid_count += __popc(result_mask);
            }

            // store the output value, one per thread
            if (output_is_valid) {
                output.element<OutputType>(i) = (is_count_op)? count: val_index;
            } else {
                output.element<OutputType>(i) = -1;
            }

            // process next element 
            i += stride;
            active_threads = __ballot_sync(active_threads, i < input.size());
        }

        // sum the valid counts across the whole block  
        size_type block_valid_count = 
            cudf::experimental::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

        if(threadIdx.x == 0) {
            atomicAdd(output_valid_count, block_valid_count);
        }
    }

struct rolling_window_launcher
{
    template<typename InputType, typename agg_op, aggregation::Kind op, typename WindowIterator, bool is_count_op=false>
    void launch(column_view const& input,
                 mutable_column_view& output,
                 WindowIterator preceding_window_begin,
                 WindowIterator following_window_begin,
                 size_type min_periods,
                 hipStream_t stream) 
        {
        if ((op == aggregation::ARGMIN) or (op == aggregation::ARGMAX))
        {
            std::cout <<"RGSL : Argmax and ArgMin"<<std::endl;
        }
        constexpr cudf::size_type block_size = 256;
        cudf::experimental::detail::grid_1d grid(input.size(), block_size);

        auto input_device_view = column_device_view::create(input, stream);
        auto output_device_view = mutable_column_device_view::create(output, stream);

        rmm::device_scalar<size_type> device_valid_count{0, stream};


        if (input.has_nulls()) {
                gpu_rolling<InputType, target_type_t<InputType, op>, agg_op, op, block_size, is_count_op, true><<<grid.num_blocks, block_size, 0, stream>>>
                    (*input_device_view, *output_device_view, device_valid_count.data(),
                     preceding_window_begin, following_window_begin, min_periods, agg_op::template identity<InputType>());
        } else {
                gpu_rolling<InputType, target_type_t<InputType, op>, agg_op, op, block_size, is_count_op, false><<<grid.num_blocks, block_size, 0, stream>>>
                    (*input_device_view, *output_device_view, device_valid_count.data(),
                     preceding_window_begin, following_window_begin, min_periods, agg_op::template identity<InputType>());
        }

        //output.set_null_count(output.size() - device_valid_count.value(stream));
    }

  template<typename T, typename agg_op, aggregation::Kind op, typename WindowIterator,
    std::enable_if_t<cudf::detail::is_supported<T, agg_op,
                                                 op == aggregation::MEAN>()>* = nullptr>
  std::unique_ptr<column> dispatch_aggregation_type(column_view const& input,
                                                    WindowIterator preceding_window_begin,
                                                    WindowIterator following_window_begin,
                                                    size_type min_periods,
                                                    rmm::mr::device_memory_resource *mr,
                                                    hipStream_t stream)
  {
    std::cout <<"dispatch_aggregation_type common"<<std::endl;
    if (input.is_empty()) return empty_like(input);

    cudf::nvtx::range_push("CUDF_ROLLING_WINDOW", cudf::nvtx::color::ORANGE);

    auto output = make_fixed_width_column(target_type(input.type(), op), input.size(),
          UNINITIALIZED, stream, mr);

    cudf::mutable_column_view output_view = output->mutable_view();
    launch<T, agg_op, op, WindowIterator>(input, output_view, preceding_window_begin,
                                                following_window_begin, min_periods, stream);

    // check the stream for debugging
    CHECK_CUDA(stream);

    cudf::nvtx::range_pop();

    return output;
  }

  template<typename T, typename agg_op, aggregation::Kind op, typename WindowIterator,
    std::enable_if_t<cudf::detail::is_string_supported<T, op>()>* = nullptr>
  std::unique_ptr<column> dispatch_aggregation_type(column_view const& input,
                                                    WindowIterator preceding_window_begin,
                                                    WindowIterator following_window_begin,
                                                    size_type min_periods,
                                                    rmm::mr::device_memory_resource *mr,
                                                    hipStream_t stream)
  {
    std::cout <<"string type"<<std::endl;
    auto output = make_numeric_column(cudf::data_type{cudf::experimental::type_to_id<size_type>()},
            input.size(), cudf::UNINITIALIZED, stream, mr);

    cudf::mutable_column_view output_view = output->mutable_view();
    if(op == aggregation::MIN) {
        launch<T, agg_op, aggregation::ARGMIN, WindowIterator>(input, output_view, preceding_window_begin,
                                                    following_window_begin, min_periods, stream);
    } else if(op == aggregation::MAX) {
        launch<T, agg_op, aggregation::ARGMAX, WindowIterator>(input, output_view, preceding_window_begin,
                                                    following_window_begin, min_periods, stream);
    } else {
        launch<T, agg_op, aggregation::COUNT, WindowIterator, true>(input, output_view, preceding_window_begin,
                                                    following_window_begin, min_periods, stream);
    }

    // check the stream for debugging
    CHECK_CUDA(stream);

    cudf::nvtx::range_pop();

    // If aggregation operation is MIN or MAX, then the output we got is a scatter map
    if((op == aggregation::MIN) or (op == aggregation::MAX)) {
        auto output_table = detail::gather(table_view{{input}}, output->view(), false, true, false, mr, stream);
        return std::make_unique<cudf::column>(std::move(output_table->get_column(0)));;
    }
    
    return output;
  }

  /**
   * @brief If we cannot perform aggregation on this type then throw an error
   */
  template<typename T, typename agg_op, aggregation::Kind op, typename WindowIterator,
    std::enable_if_t<!cudf::detail::is_supported<T, agg_op,
                                                 op == aggregation::MEAN>() and !cudf::detail::is_string_supported<T, op>()>* = nullptr>
  std::unique_ptr<column> dispatch_aggregation_type(column_view const& input,
                                                    WindowIterator preceding_window_begin,
                                                    WindowIterator following_window_begin,
                                                    size_type min_periods,
                                                    rmm::mr::device_memory_resource *mr,
                                                    hipStream_t stream)
  {
    CUDF_FAIL("Unsupported column type/operation combo. Only `min` and `max` are supported for "
              "non-arithmetic types for aggregations.");
  }

  /**
   * @brief Helper function for gdf_rolling. Deduces the type of the
   * aggregation column and type and calls another function to invoke the
   * rolling window kernel.
   */
  template <typename T, typename WindowIterator>
  std::unique_ptr<column> operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& aggr,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream)
  {
    switch (aggr->kind) {
    case aggregation::SUM:
      return dispatch_aggregation_type<T, corresponding_operator<aggregation::SUM>::type,
                                       aggregation::SUM>(input, preceding_window_begin,
                                                               following_window_begin, min_periods,
                                                               mr, stream);
    case aggregation::MIN:
      return dispatch_aggregation_type<T, corresponding_operator<aggregation::MIN>::type,
                                       aggregation::MIN>(input, preceding_window_begin,
                                                              following_window_begin, min_periods,
                                                              mr, stream);
    case aggregation::MAX:
      return dispatch_aggregation_type<T, corresponding_operator<aggregation::MAX>::type,
                                       aggregation::MAX>(input, preceding_window_begin,
                                                              following_window_begin, min_periods,
                                                              mr, stream);
    case aggregation::COUNT:
      // for count, use size_type rather than the input type (never load the input)
      return dispatch_aggregation_type<cudf::size_type, cudf::DeviceCount,
                                       aggregation::COUNT>(input, preceding_window_begin,
                                                                following_window_begin, min_periods,
                                                                mr, stream);
    case aggregation::MEAN:
      return dispatch_aggregation_type<T, corresponding_operator<aggregation::SUM>::type,
                                       aggregation::MEAN>(input, preceding_window_begin,
                                                               following_window_begin, min_periods,
                                                               mr, stream);
    default:
      // TODO: need a nice way to convert enums to strings, same would be useful for groupby
      CUDF_FAIL("Rolling aggregation function not implemented");
    }
  }
};

} // namespace anonymous

// Applies a rolling window function to the values in a column.
template <typename WindowIterator>
std::unique_ptr<column> rolling_window(column_view const& input,
                                       WindowIterator preceding_window_begin,
                                       WindowIterator following_window_begin,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& aggr,
                                       rmm::mr::device_memory_resource* mr,
                                       hipStream_t stream = 0)
{
  return cudf::experimental::type_dispatcher(input.type(),
                                             rolling_window_launcher{},
                                             input, preceding_window_begin, following_window_begin,
                                             min_periods, aggr, mr, stream);
}

} // namespace detail

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& aggr,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS((preceding_window >= 0) && (following_window >= 0) && (min_periods >= 0),
               "Window sizes and min periods must be non-negative");

  auto preceding_window_begin = thrust::make_constant_iterator(preceding_window);
  auto following_window_begin = thrust::make_constant_iterator(following_window);

  return cudf::experimental::detail::rolling_window(input, preceding_window_begin,
                                                    following_window_begin, min_periods, aggr, mr, 0);
}

// Applies a variable-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& preceding_window,
                                       column_view const& following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& aggr,
                                       rmm::mr::device_memory_resource* mr)
{
  if (preceding_window.size() == 0 || following_window.size() == 0) return empty_like(input);

  CUDF_EXPECTS(preceding_window.type().id() == INT32 && following_window.type().id() == INT32,
               "preceding_window/following_window must have INT32 type");

  CUDF_EXPECTS(preceding_window.size() == input.size() && following_window.size() == input.size(),
               "preceding_window/following_window size must match input size");

  return cudf::experimental::detail::rolling_window(input, preceding_window.begin<size_type>(),
                                                    following_window.begin<size_type>(),
                                                    min_periods, aggr, mr, 0);
}

} // namespace experimental 
} // namespace cudf
