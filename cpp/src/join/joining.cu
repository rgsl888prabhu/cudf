#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include <cudf/types.hpp>
#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <utilities/column_utils.hpp>
#include <utilities/error_utils.hpp>
#include <utilities/type_dispatcher.hpp>
#include <utilities/nvtx/nvtx_utils.h>
#include <string/nvcategory_util.hpp>
#include <nvstrings/NVCategory.h>
#include <copying/gather.hpp>
#include "joining.h"

#include <limits>
#include <set>
#include <vector>

// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = gdf_index_type;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a repeated value
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @param[in] value The value to be filled into the buffer
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocValueBuffer(data_type ** buffer,
                           const gdf_size_type buffer_length,
                           const data_type value) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::fill(thrust::device, *buffer, *buffer + buffer_length, value);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a sequence
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocSequenceBuffer(data_type ** buffer,
                              const gdf_size_type buffer_length) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::sequence(thrust::device, *buffer, *buffer + buffer_length);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Trivially computes full join of two tables if one of the tables
 * are empty
 * 
 * @param[in] left_size The size of the left table
 * @param[in] right_size The size of the right table
 * @param[in] rightcol The right set of columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 */
/* ----------------------------------------------------------------------------*/
void trivial_full_join(
        const gdf_size_type left_size,
        const gdf_size_type right_size,
        gdf_column *left_result,
        gdf_column *right_result) {
    // Deduce the type of the output gdf_columns
    gdf_dtype dtype;
    switch(sizeof(output_index_type))
    {
      case 1 : dtype = GDF_INT8;  break;
      case 2 : dtype = GDF_INT16; break;
      case 4 : dtype = GDF_INT32; break;
      case 8 : dtype = GDF_INT64; break;
    }

    output_index_type *l_ptr{nullptr};
    output_index_type *r_ptr{nullptr};
    gdf_size_type result_size{0};
    CUDF_EXPECTS((left_size != 0) || (right_size != 0), "Dataset is empty");
    if (left_size == 0) {
        allocValueBuffer(&l_ptr, right_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&r_ptr, right_size);
        result_size = right_size;
    } else if (right_size == 0) {
        allocValueBuffer(&r_ptr, left_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&l_ptr, left_size);
        result_size = left_size;
    }
    gdf_column_view( left_result, l_ptr, nullptr, result_size, dtype);
    gdf_column_view(right_result, r_ptr, nullptr, result_size, dtype);
    CUDA_CHECK_LAST();
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Computes the join operation between two sets of columns
 * 
 * @param[in] num_cols The number of columns to join
 * @param[in] leftcols  cudf table of left columns to join
 * @param[in] rightcols cudf table of right  columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 * @param[in] join_context A structure that determines various run parameters, such as
 *                         whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @returns void
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
void join_call( int num_cols, cudf::table leftcols, cudf::table rightcols,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{
  CUDF_EXPECTS( 0 != num_cols, "Dataset is empty");
  CUDF_EXPECTS( 0 != leftcols.num_columns(), "Left Dataset is empty");
  //CUDF_EXPECTS( 0 != leftcols.num_rows(), "Left Dataset is empty");
  CUDF_EXPECTS( 0 != rightcols.num_columns(), "Right Dataset is empty");
  //CUDF_EXPECTS( 0 != rightcols.num_rows(), "Right Dataset is empty");
  CUDF_EXPECTS( nullptr != join_context, "Invalid join context");

  const auto left_col_size = leftcols.get_column(0)->size;
  const auto right_col_size = rightcols.get_column(0)->size;
  
  CUDF_EXPECTS( left_col_size < MAX_JOIN_SIZE, "left column size is too big");
  CUDF_EXPECTS( right_col_size < MAX_JOIN_SIZE, "right column size is too big");


  // If both frames are empty, return immediately
  if((0 == left_col_size ) && (0 == right_col_size)) {
    return;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)){
    return;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return;
  }

  // If Full Join and either table is empty, compute trivial full join
  if( (JoinType::FULL_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    trivial_full_join(left_col_size, right_col_size, left_result, right_result);
    return;  
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < num_cols; i++) {
    CUDF_EXPECTS (!((left_col_size > 0) && (nullptr == leftcols.get_column(i)->data)), "One of the column is null left column set");
    CUDF_EXPECTS (!((right_col_size > 0) && (nullptr == rightcols.get_column(i)->data)), "One of the column is null in right column set");
    CUDF_EXPECTS (rightcols.get_column(i)->dtype == leftcols.get_column(i)->dtype, "DTYPE mismatch");
    CUDF_EXPECTS (left_col_size == leftcols.get_column(i)->size, "left column size mismatch");
    CUDF_EXPECTS (right_col_size == rightcols.get_column(i)->size, "right column size mismatch");

    // Ensure GDF_TIMESTAMP columns have the same resolution
    if (GDF_TIMESTAMP == rightcols.get_column(i)->dtype) {
      CUDF_EXPECTS(
          rightcols.get_column(i)->dtype_info.time_unit == leftcols.get_column(i)->dtype_info.time_unit,
          "Timestamp resolution mis-match");
    }
  }

  gdf_method join_method = join_context->flag_method; 
  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_JOIN", JOIN_COLOR);

  switch(join_method)
  {
    case GDF_HASH:
      {
        gdf_error_code = join_hash<join_type, output_index_type>(leftcols, rightcols, left_result, right_result);
        CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        break;
      }
    case GDF_SORT:
      {
        // Sort based joins only support single column joins
        if(1 == num_cols)
        {
          gdf_error_code =  sort_join<join_type, output_index_type>(leftcols.get_column(0), rightcols.get_column(0), left_result, right_result);
          CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        }
        else
        {
          CUDF_EXPECTS(false, "Too many columns to join");
        }

        break;
      }
    default:
      CUDF_EXPECTS(false, "Unsupported Method");
  }

  POP_RANGE();
}

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> construct_join_output_df(
        cudf::table & ljoincols,
        cudf::table & rjoincols,
        cudf::table & left_cols, 
        std::vector const& left_j_cols,
        cudf::table & right_cols,
        gdf_column * left_indices,
        gdf_column * right_indices,
        std::vector const& left_join_result_cols,
        std::vector const& right_join_result_cols,
        int num_cols_joined_result) {

  PUSH_RANGE("LIBGDF_JOIN_OUTPUT", JOIN_COLOR);
    //create left and right input table with columns not joined on
    int num_left_cols = left_cols.num_columns();
    int num_right_cols = right_cols.num_columns();
    int result_num_cols = num_left_cols + num_right_cols - num_cols_joined_result;

    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;
    for (int i = 0; i < num_left_cols; ++i) {
        if (std::find(left_join_result_cols.begin(), left_join_result_cols.end(), i)
            == left_join_result_cols.end()) {
            lnonjoincol.push_back(left_cols.get_column(i));
        }
    }
    for (int i = 0; i < num_right_cols; ++i) {
        if (std::find(right_join_result_cols.begin(), right_join_result_cols.end(), i) 
            == right_join_result_cols.end()) {
            rnonjoincol.push_back(right_cols.get_column(i));
        }
    }
    //TODO : Invalid api

    gdf_size_type join_size = left_indices->size;

    std::vector <gdf_dtype> ldtypes;
    std::vector <gdf_dtype> rdtypes;
    
    std::transform (left_cols.begin(), left_cols.end(), ldtypes.being(), [ ](gdf_column* col, gdf_dtypes dtypes) {
      dtypes = col->dtypes};
    )
    std::transform (right_cols.begin(), right_cols.end(), rdtypes.being(), [ ](gdf_column* col, gdf_dtypes dtypes) {
      dtypes = col->dtypes};
    )

    cudf::table result_left(join_size, ldtypes, true)
    cudf::table result_right(join_size, rdtypes, true)
 
    std::vector<gdf_column*> result_lnonjoincol;
    std::vector<gdf_column*> result_rnonjoincol;
    std::vector<gdf_column*> result_joincol;

    for (int lindex = 0, lindex < num_left_cols; lindex++)
    {
        // Accumalate the left non-join col
        if (std::find(left_join_result_cols.begin(), left_join_result_cols.end(), lindex++)
            == left_join_result_cols.end()) {
            result_lnonjoincol.push_back(result_left.get_colun(lindex))
        }
        // Accumalate the join-col 
        else
        {
            result_joincol.push_back(result_left.get_colun(lindex))
        }
    }
    
    // Accumalate the right non-join col
    for (int rindex = 0, rindex < num_left_cols; rindex++)
    {
        if (std::find(right_join_result_cols.begin(), right_join_result_cols.end(), rindex)
            == right_join_result_cols.end()) {
            result_rnonjoincol.push_back(result_right.get_colun(rindex))
        }
    }
 
    bool const check_bounds{ join_type != JoinType::INNER_JOIN };

    // Construct the left columns
    if (0 != lnonjoincol.size()) {
      cudf::table left_source_table(lnonjoincol.data(), lnonjoincol.size());
      cudf::table left_destination_table(result_lnonjoincol.data(), result_lnonjoincol.size());

      cudf::detail::gather(&left_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &left_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(left_source_table,left_destination_table);
      CHECK_STREAM(0);
      GDF_REQUIRE(update_err == GDF_SUCCESS,update_err);
    }

    // Construct the right columns
    if (0 != rnonjoincol.size()) {
      cudf::table right_source_table(rnonjoincol.data(), rnonjoincol.size());
      cudf::table right_destination_table(result_rnonjoincol.data(), result_rnonjoincol.size());

      cudf::detail::gather(&right_source_table,
                           static_cast<index_type const *>(right_indices->data),
                           &right_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(right_source_table,right_destination_table);
      CHECK_STREAM(0);
      GDF_REQUIRE(update_err == GDF_SUCCESS,update_err);
    }

    // Construct the joined columns
    if (0 != ljoincol.size() && num_cols_joined_result > 0) {

      std::vector <gdf_column *> l_join(num_cols_joined_result);
      std::vector <gdf_column *> r_join(num_cols_joined_result);
      for (int join_ind = 0; join_ind < num_cols_joined_result; ++join_ind)
      {
          std::vector<int>::iterator itr = std::find(left_j_cols.begin(), left_j_cols.end(),
               left_join_result_cols[join_ind]);

          l_join[join_ind] =
               ljoincol[std::distance(left_j_cols.begin(), itr)];

          r_join[join_ind] =
               rjoincol[std::distance(left_j_cols.begin(), itr)];
      }
      cudf::table join_source_table(l_join.data(), l_join.size());
      cudf::table join_destination_table(result_joincol.data(),
                                         result_joincol.size());

      // Gather valid rows from the right table
      // TODO: Revisit this, because it probably can be done more efficiently
      if (JoinType::FULL_JOIN == join_type) {
        cudf::table right_source_table(r_join.data(), r_join.size());

        cudf::detail::gather(
            &right_source_table,
            static_cast<index_type const *>(right_indices->data),
            &join_destination_table, check_bounds);
        CHECK_STREAM(0);
      }

      cudf::detail::gather(&join_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &join_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(join_source_table,join_destination_table);
      CHECK_STREAM(0);
      GDF_REQUIRE(update_err == GDF_SUCCESS,update_err);
    }
    
    POP_RANGE();
    return std::pair<cudf::table, cudf::table>(result_left, result_right)
}

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> join_call_compute_df(
                         cudf::table const& left_cols, 
                         std::vector <index_type> left_join_cols,
                         cudf::table const& right_cols,
                         std::vector <index_type> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <index_type> left_join_result_cols,
                         std::vector <index_type> right_join_result_cols) {

  int num_left_cols = left_cols.num_columns();
  int num_right_cols = right_cols.num_columns();
  int num_cols_to_join = left_join_cols.size();

  CUDF_EXPECTS (0 != num_left_cols, "Left table is empty")
  CUDF_EXPECTS (0 != num_right_cols, "Right table is empty")
  CUDF_EXPECTS (nullptr != join_context, "Join context is invalid");

  if (0 == num_cols_to_join):
  {
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {})
  }

  for(int column_index = 0; column_index  < num_left_cols; column_index++){
    CUDF_EXPECTS(left_cols.get_column(column_index)->dtype != GDF_invalid, "Unsupported Dtype in Left column");
  }
  for(int column_index = 0; column_index  < num_right_cols; column_index++){
    CUDF_EXPECTS(right_cols.get_column(column_index)->dtype != GDF_invalid, "Unsupported Dtype in right column");
  }

  auto const left_col_size = left_cols.get_column(0)->size;
  auto const right_col_size = right_cols.get_column(0)->size;

  // If the inputs are empty, immediately return
  if ((0 == left_col_size) && (0 == right_col_size)) {
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {})
  }

  // If left join and the left table is empty, return immediately
  if ((JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)) {
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {})
  }

  // If Inner Join and either table is empty, return immediately
  if ((JoinType::INNER_JOIN == join_type) &&
      ((0 == left_col_size) || (0 == right_col_size))) {
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {})
  }


  //if the inputs are nvcategory we need to make the dictionaries comparable
  bool at_least_one_category_column = false;
  for(int join_column_index = 0; join_column_index < num_cols_to_join; join_column_index++){
    at_least_one_category_column |= left_cols.get_column(left_join_cols[join_column_index])->dtype == GDF_STRING_CATEGORY;
  }

  std::vector<gdf_column*> new_left_cols(left_cols.begin(), left_cols.end());
  std::vector<gdf_column*> new_right_cols(right_cols.begin(), right_cols.end());
  std::vector<gdf_column *> temp_columns_to_free;
  if(at_least_one_category_column){
    for(int join_column_index = 0; join_column_index < num_cols_to_join; join_column_index++){
      if(left_cols[left_join_cols[join_column_index]]->dtype == GDF_STRING_CATEGORY){
        GDF_REQUIRE(right_cols[right_join_cols[join_column_index]]->dtype == GDF_STRING_CATEGORY, GDF_DTYPE_MISMATCH);

        gdf_column * left_original_column = new_left_cols[left_join_cols[join_column_index]];
        gdf_column * right_original_column = new_right_cols[right_join_cols[join_column_index]];




        gdf_column * new_left_column_ptr = new gdf_column{};
        gdf_column * new_right_column_ptr = new gdf_column{};

        temp_columns_to_free.push_back(new_left_column_ptr);
        temp_columns_to_free.push_back(new_right_column_ptr);


        gdf_column * input_join_columns_merge[2] = {left_original_column, right_original_column};
        gdf_column * new_join_columns[2] = {new_left_column_ptr,
            new_right_column_ptr};
        gdf_column_view(new_left_column_ptr, nullptr, nullptr, left_original_column->size, GDF_STRING_CATEGORY);
        gdf_column_view(new_right_column_ptr, nullptr, nullptr, right_original_column->size, GDF_STRING_CATEGORY);

        int col_width = cudf::byte_width(*new_left_column_ptr);
        RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->data), col_width * left_original_column->size, 0) ); // TODO: non-default stream?
        if(left_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(left_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_left_column_ptr->valid, left_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(left_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_left_column_ptr->valid = nullptr;
        }
        new_left_column_ptr->null_count = left_original_column->null_count;


        RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->data), col_width * right_original_column->size, 0) ); // TODO: non-default stream?
        if(right_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(right_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_right_column_ptr->valid, right_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(right_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_right_column_ptr->valid = nullptr;
        }
        new_right_column_ptr->null_count = right_original_column->null_count;
        gdf_error err = sync_column_categories(input_join_columns_merge,
            new_join_columns,
            2);

        GDF_REQUIRE(GDF_SUCCESS == err, err);

        new_left_cols[left_join_cols[join_column_index]] = new_join_columns[0];
        new_right_cols[right_join_cols[join_column_index]] = new_join_columns[1];
        CHECK_STREAM(0);
      }
    }
  }

  cudf::table  updated_left_cols(new_left_cols);
  cudf::table  updated_right_cols(new_right_cols);
  

  // If index outputs are not requested, create columns to store them
  // for computing combined join output
  gdf_column *left_index_out = left_indices;
  gdf_column *right_index_out = right_indices;

  using gdf_col_pointer =
      typename std::unique_ptr<gdf_column, std::function<void(gdf_column *)>>;
  auto gdf_col_deleter = [](gdf_column *col) {
    col->size = 0;
    if (col->data) {
      RMM_FREE(col->data, 0);
    }
    if (col->valid) {
      RMM_FREE(col->valid, 0);
    }
  };
  gdf_col_pointer l_index_temp, r_index_temp;

  if (nullptr == left_indices) {
    l_index_temp = {new gdf_column{}, gdf_col_deleter};
    left_index_out = l_index_temp.get();
    }

    if (nullptr == right_indices) {
        r_index_temp = {new gdf_column{}, gdf_col_deleter};
        right_index_out = r_index_temp.get();
    }

    //get column pointers to join on
    std::vector<gdf_column*> ljoincol;
    std::vector<gdf_column*> rjoincol;
    for (int i = 0; i < num_cols_to_join; ++i) {
        ljoincol.push_back(left_cols.get_column(left_join_cols[i]));
        rjoincol.push_back(right_cols.get_column(right_join_cols[i]));
    }


    cudf::table ljoin_cols(ljoincol);
    cudf::table rjoin_cols(rjoincol);
    join_call<join_type>(num_cols_to_join,
            ljoin_cols, rjoin_cols,
            left_index_out, right_index_out,
            join_context);
    CHECK_STREAM(0);

    std::pair<cudf::table, cudf::table> merged_result =
        construct_join_output_df<join_type, index_type>(
            ljoincol, rjoincol,
            left_cols, num_left_cols, left_join_cols,
            right_cols, num_right_cols, right_join_cols,
            num_cols_to_join, result_num_cols, result_cols,
            left_index_out, right_index_out, left_join_result_cols, 
            right_join_result_cols, num_cols_joined_result);
    CHECK_STREAM(0);
    l_index_temp.reset(nullptr);
    r_index_temp.reset(nullptr);





    //freeing up the temp column used to synch categories between columns
    for(unsigned int column_to_free = 0; column_to_free < temp_columns_to_free.size(); column_to_free++){
      gdf_column_free(temp_columns_to_free[column_to_free]);
      delete temp_columns_to_free[column_to_free];
    }

    CHECK_STREAM(0);


    return merged_result;
}

std::pair<cudf::table, cudf::table> gdf_left_join(
                         cudf::table const& left_cols,
                         std::vector <index_type> left_join_cols,
                         cudf::table const& right_cols,
                         std::vector <index_type> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <index_type> left_join_result_cols,
                         std::vector <index_type> right_join_result_cols) {
    return join_call_compute_df<JoinType::LEFT_JOIN, output_index_type>(
                     left_cols, 
                     left_join_cols,
                     right_cols,
                     right_join_cols,
                     left_indices,
                     right_indices,
                     join_context,
                     left_join_result_cols,
                     right_join_result_cols,
                     num_cols_joined_result);
}

std::pair<cudf::table, cudf::table> gdf_inner_join(
                         cudf::table const& left_cols,
                         std::vector <index_type> left_join_cols,
                         cudf::table const& right_cols,
                         std::vector <index_type> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <index_type> left_join_result_cols,
                         std::vector <index_type> right_join_result_cols) {
    return join_call_compute_df<JoinType::INNER_JOIN, output_index_type>(
                     left_cols,
                     left_join_cols,
                     right_cols,
                     right_join_cols,
                     left_indices,
                     right_indices,
                     join_context,
                     left_join_result_cols,
                     right_join_result_cols,
                     num_cols_joined_result);
}

std::pair<cudf::table, cudf::table> gdf_full_join(
                         cudf::table const& left_cols,
                         std::vector <index_type> left_join_cols,
                         cudf::table const& right_cols,
                         std::vector <index_type> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <index_type> left_join_result_cols,
                         std::vector <index_type> right_join_result_cols) {
    return join_call_compute_df<JoinType::FULL_JOIN, output_index_type>(
                     left_cols,
                     left_join_cols,
                     right_cols,
                     right_join_cols,
                     left_indices,
                     right_indices,
                     join_context,
                     left_join_result_cols,
                     right_join_result_cols,
                     num_cols_joined_result);
}
