#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include <cudf/types.hpp>
#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <utilities/column_utils.hpp>
#include <utilities/error_utils.hpp>
#include <utilities/type_dispatcher.hpp>
#include <utilities/nvtx/nvtx_utils.h>
#include <string/nvcategory_util.hpp>
#include <nvstrings/NVCategory.h>
#include <copying/gather.hpp>
#include "joining.h"

#include <limits>
#include <set>
#include <vector>

// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = gdf_index_type;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

namespace cudf {
/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a repeated value
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @param[in] value The value to be filled into the buffer
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocValueBuffer(data_type ** buffer,
                           const gdf_size_type buffer_length,
                           const data_type value) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::fill(thrust::device, *buffer, *buffer + buffer_length, value);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a sequence
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocSequenceBuffer(data_type ** buffer,
                              const gdf_size_type buffer_length) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::sequence(thrust::device, *buffer, *buffer + buffer_length);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Trivially computes full join of two tables if one of the tables
 * are empty
 * 
 * @param[in] left_size The size of the left table
 * @param[in] right_size The size of the right table
 * @param[in] rightcol The right set of columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 */
/* ----------------------------------------------------------------------------*/
void trivial_full_join(
        const gdf_size_type left_size,
        const gdf_size_type right_size,
        gdf_column *left_result,
        gdf_column *right_result) {
    // Deduce the type of the output gdf_columns
    gdf_dtype dtype;
    switch(sizeof(output_index_type))
    {
      case 1 : dtype = GDF_INT8;  break;
      case 2 : dtype = GDF_INT16; break;
      case 4 : dtype = GDF_INT32; break;
      case 8 : dtype = GDF_INT64; break;
    }

    output_index_type *l_ptr{nullptr};
    output_index_type *r_ptr{nullptr};
    gdf_size_type result_size{0};
    CUDF_EXPECTS((left_size != 0) || (right_size != 0), "Dataset is empty");
    if (left_size == 0) {
        allocValueBuffer(&l_ptr, right_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&r_ptr, right_size);
        result_size = right_size;
    } else if (right_size == 0) {
        allocValueBuffer(&r_ptr, left_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&l_ptr, left_size);
        result_size = left_size;
    }
    gdf_column_view( left_result, l_ptr, nullptr, result_size, dtype);
    gdf_column_view(right_result, r_ptr, nullptr, result_size, dtype);
    CUDA_CHECK_LAST();
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Computes the join operation between two sets of columns
 * 
 * @param[in] num_cols The number of columns to join
 * @param[in] leftcols  cudf table of left columns to join
 * @param[in] rightcols cudf table of right  columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 * @param[in] join_context A structure that determines various run parameters, such as
 *                         whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @returns void
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
void join_call( int num_cols, cudf::table leftcols, cudf::table rightcols,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{
  CUDF_EXPECTS( 0 != num_cols, "Dataset is empty");
  CUDF_EXPECTS( 0 != leftcols.num_columns(), "Left Dataset is empty");
  //CUDF_EXPECTS( 0 != leftcols.num_rows(), "Left Dataset is empty");
  CUDF_EXPECTS( 0 != rightcols.num_columns(), "Right Dataset is empty");
  //CUDF_EXPECTS( 0 != rightcols.num_rows(), "Right Dataset is empty");
  CUDF_EXPECTS( nullptr != join_context, "Invalid join context");

  const auto left_col_size = leftcols.get_column(0)->size;
  const auto right_col_size = rightcols.get_column(0)->size;
  
  CUDF_EXPECTS( left_col_size < MAX_JOIN_SIZE, "left column size is too big");
  CUDF_EXPECTS( right_col_size < MAX_JOIN_SIZE, "right column size is too big");


  // If both frames are empty, return immediately
  if((0 == left_col_size ) && (0 == right_col_size)) {
    return;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)){
    return;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return;
  }

  // If Full Join and either table is empty, compute trivial full join
  if( (JoinType::FULL_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    trivial_full_join(left_col_size, right_col_size, left_result, right_result);
    return;  
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < num_cols; i++) {
    CUDF_EXPECTS (!((left_col_size > 0) && (nullptr == leftcols.get_column(i)->data)), "One of the column is null left column set");
    CUDF_EXPECTS (!((right_col_size > 0) && (nullptr == rightcols.get_column(i)->data)), "One of the column is null in right column set");
    CUDF_EXPECTS (rightcols.get_column(i)->dtype == leftcols.get_column(i)->dtype, "DTYPE mismatch");
    CUDF_EXPECTS (left_col_size == leftcols.get_column(i)->size, "left column size mismatch");
    CUDF_EXPECTS (right_col_size == rightcols.get_column(i)->size, "right column size mismatch");

    // Ensure GDF_TIMESTAMP columns have the same resolution
    if (GDF_TIMESTAMP == rightcols.get_column(i)->dtype) {
      CUDF_EXPECTS(
          rightcols.get_column(i)->dtype_info.time_unit == leftcols.get_column(i)->dtype_info.time_unit,
          "Timestamp resolution mis-match");
    }
  }

  gdf_method join_method = join_context->flag_method; 
  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_JOIN", JOIN_COLOR);

  switch(join_method)
  {
    case GDF_HASH:
      {
        gdf_error_code = join_hash<join_type, output_index_type>(leftcols, rightcols, left_result, right_result);
        CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        break;
      }
    case GDF_SORT:
      {
        // Sort based joins only support single column joins
        if(1 == num_cols)
        {
          gdf_error_code =  sort_join<join_type, output_index_type>(leftcols.get_column(0), rightcols.get_column(0), left_result, right_result);
          CUDF_EXPECTS(gdf_error_code == GDF_SUCCESS, "GDF Error");
        }
        else
        {
          CUDF_EXPECTS(false, "Too many columns to join");
        }

        break;
      }
    default:
      CUDF_EXPECTS(false, "Unsupported Method");
  }

  POP_RANGE();
}

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> construct_join_output_df(
        cudf::table & ljoincols,
        cudf::table & rjoincols,
        cudf::table & left_cols, 
        std::vector<int> & left_j_cols,
        cudf::table & right_cols,
        gdf_column * left_indices,
        gdf_column * right_indices,
        std::vector<int> const& left_join_result_cols,
        std::vector<int> const& right_join_result_cols) {


  PUSH_RANGE("LIBGDF_JOIN_OUTPUT", JOIN_COLOR);
    //create left and right input table with columns not joined on
    int num_left_cols = left_cols.num_columns();
    int num_right_cols = right_cols.num_columns();
    int num_cols_joined_result = left_join_result_cols.size();
    std::cout<<"num_left_cols "<<num_left_cols<<std::endl;
    std::cout<<"num_right_cols "<<num_right_cols<<std::endl;
    std::cout<<"num_cols_joined_result "<<num_cols_joined_result<<std::endl;
    std::cout<<"ljoincols "<<ljoincols.num_columns()<<std::endl;
    std::cout<<"rjoincols "<<rjoincols.num_columns()<<std::endl;
    // int result_num_cols = num_left_cols + num_right_cols - num_cols_joined_result;

    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;
    for (int i = 0; i < num_left_cols; ++i) {
        if (std::find(left_join_result_cols.begin(), left_join_result_cols.end(), i)
            == left_join_result_cols.end()) {
            lnonjoincol.push_back(left_cols.get_column(i));
        }
    }
    for (int i = 0; i < num_right_cols; ++i) {
        if (std::find(right_join_result_cols.begin(), right_join_result_cols.end(), i) 
            == right_join_result_cols.end()) {
            rnonjoincol.push_back(right_cols.get_column(i));
        }
    }
    //TODO : Invalid api

    gdf_size_type join_size = left_indices->size;
    std::cout<<"join_size "<<join_size<<std::endl;
    std::cout<<"Left cols size " <<lnonjoincol.size()<<std::endl;
    std::cout<<"Right cols size " <<rnonjoincol.size()<<std::endl;

    std::vector <gdf_dtype> ldtypes;
    std::vector <gdf_dtype> rdtypes;
    
    //std::transform (left_cols.begin(), left_cols.end(), ldtypes.begin(), [ ](gdf_column* col) {
    //  return col->dtype;}
    //);
    for (int i=0; i < num_left_cols; ++i)
    {
        ldtypes.push_back(left_cols.get_column(i)->dtype);
    }

    for (int i=0; i < num_right_cols; ++i)
    {
        if (std::find(right_join_result_cols.begin(), right_join_result_cols.end(), i) 
            == right_join_result_cols.end()) {
            rdtypes.push_back(right_cols.get_column(i)->dtype);
        }

    }
    
    std::cout<<"RGSL : Size of rdtypes"<<rdtypes.size()<<std::endl;
    std::cout<<"RGSL : Size of ldtypes"<<ldtypes.size()<<std::endl;

    cudf::table result_left(join_size, ldtypes, true);
    cudf::table result_right(join_size, rdtypes, true);
    
    std::cout<<"The size of result left "<<result_left.num_columns()<<std::endl;
    std::cout<<"The size of result right "<<result_right.num_columns()<<std::endl;
 
    std::vector<gdf_column*> result_lnonjoincol;
    std::vector<gdf_column*> result_rnonjoincol;
    std::vector<gdf_column*> result_joincol;

    std::cout<<"Before left"<<std::endl;
    for (int lindex = 0; lindex < num_left_cols; ++lindex)
    {
        // Accumalate the left non-join col
        if (std::find(left_join_result_cols.begin(), left_join_result_cols.end(), lindex)
            == left_join_result_cols.end()) {
            std::cout<<"result left col address "<<result_left.get_column(lindex)<<std::endl;
            result_lnonjoincol.push_back(result_left.get_column(lindex));
        }
        // Accumalate the join-col 
        else
        {
            std::cout<<"Found one to join "<<lindex<<std::endl;
            std::cout<<"result join_col address "<<result_left.get_column(lindex)<<std::endl;
            result_joincol.push_back(result_left.get_column(lindex));
        }
    }
    
    std::cout<<"Before right"<<std::endl;
    // Accumalate the right non-join col
    for (int rindex = 0; rindex < num_right_cols-num_cols_joined_result; ++rindex)
    {
            std::cout<<"result right col address "<<result_right.get_column(rindex)<<std::endl;
            result_rnonjoincol.push_back(result_right.get_column(rindex));
    }
 
    bool const check_bounds{ join_type != JoinType::INNER_JOIN };

    std::cout<<"Before left result"<<std::endl;
    // Construct the left columns
    if (0 != lnonjoincol.size()) {
    std::cout<<"inside left result"<<std::endl;
      std::cout<<"src left col address "<<lnonjoincol[0]<<std::endl;
      cudf::table left_source_table(lnonjoincol);
      cudf::table left_destination_table(result_lnonjoincol);
      std::cout <<"RGSL : left src table size " <<left_source_table.num_columns()<<std::endl;
      std::cout <<"RGSL : left des table size " <<left_destination_table.num_columns()<<std::endl;

      cudf::detail::gather(&left_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &left_destination_table, check_bounds);
      std::vector <int64_t> temp (join_size);
      for (unsigned int k=0; k < result_lnonjoincol.size(); ++k)
      {
      std::cout<<"Dtype in result "<<result_lnonjoincol[k]->dtype<<std::endl;
      std::cout<<"Dtype in src "<<lnonjoincol[k]->dtype<<std::endl;
      hipMemcpy(temp.data(), result_lnonjoincol[k]->data, sizeof(int64_t)*result_lnonjoincol[k]->size,hipMemcpyDeviceToHost);

      for (unsigned int i = 0; i < temp.size(); ++i)
      {
          std::cout<<i<<". "<<temp[i]<<std::endl;
      }
      }
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(left_source_table,left_destination_table);
      CHECK_STREAM(0);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table throwing a GDF error");
    }
    if (rnonjoincol[0] == nullptr)
        std::cout<<"RGSL : Null value for nonjoincol"<<std::endl;
    if (result_rnonjoincol[0] == nullptr)
        std::cout<<"RGSL : Null value for result"<<std::endl;

    std::cout<<"Before right result"<<std::endl;
    // Construct the right columns
    if (0 != rnonjoincol.size()) {
      std::cout<<"src right col address "<<rnonjoincol[0]<<std::endl;
      std::cout<<"inside right result"<<std::endl;
      std::cout<<"right src and dst size" <<rnonjoincol.size()<<" "<<result_rnonjoincol.size()<<std::endl;
      cudf::table right_source_table(rnonjoincol);
      cudf::table right_destination_table(result_rnonjoincol);

      std::cout <<"RGSL : right src table size " <<right_source_table.num_columns()<<std::endl;
      std::cout <<"RGSL : right des table size " <<right_destination_table.num_columns()<<std::endl;
      std::vector <int64_t> temp (join_size);
      for (unsigned int k=0; k < result_rnonjoincol.size(); ++k)
      {
      std::cout<<"Dtype in result "<<result_rnonjoincol[k]->dtype<<std::endl;
      std::cout<<"Dtype in src "<<rnonjoincol[k]->dtype<<std::endl;
      hipMemcpy(temp.data(), result_rnonjoincol[k]->data, sizeof(int64_t)*result_rnonjoincol[k]->size,hipMemcpyDeviceToHost);

      for (unsigned int i = 0; i < temp.size(); ++i)
      {
          std::cout<<i<<". "<<temp[i]<<std::endl;
      }
      }

      cudf::detail::gather(&right_source_table,
                           static_cast<index_type const *>(right_indices->data),
                           &right_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(right_source_table,right_destination_table);
      CHECK_STREAM(0);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table throwing a GDF error");
    }

    std::cout<<"Before join result"<<std::endl;
    // Construct the joined columns
    if (0 != ljoincols.num_columns() && num_cols_joined_result > 0) {
    std::cout<<"inside join result"<<std::endl;

      std::vector <gdf_column *> l_join;
      std::vector <gdf_column *> r_join;
      for (int join_ind = 0; join_ind < num_cols_joined_result; ++join_ind)
      {
          std::vector<int>::iterator itr = std::find(left_j_cols.begin(), left_j_cols.end(),
               left_join_result_cols[join_ind]);

          int index = std::distance(left_j_cols.begin(), itr);

          std::cout<<"The join index is "<< index <<std::endl;         

          l_join.push_back(ljoincols.get_column(index));

      if (JoinType::FULL_JOIN == join_type)
          r_join.push_back(rjoincols.get_column(index));
      }
      std::cout<<"src left join col address "<<l_join[0]<<std::endl;
      cudf::table join_source_table(l_join);
      cudf::table join_destination_table(result_joincol);

      // Gather valid rows from the right table
      // TODO: Revisit this, because it probably can be done more efficiently
      if (JoinType::FULL_JOIN == join_type) {
        std::cout<<"src right join col address "<<r_join[0]<<std::endl;
        cudf::table right_source_table(r_join);

        cudf::detail::gather(
            &right_source_table,
            static_cast<index_type const *>(right_indices->data),
            &join_destination_table, check_bounds);
        CHECK_STREAM(0);
      }

      cudf::detail::gather(&join_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &join_destination_table, check_bounds);
     #if 1 
      std::vector <int64_t> temp (join_size);
      for (unsigned int k=0; k < result_joincol.size(); ++k)
      {
      std::cout<<"Dtype in result "<<result_joincol[k]->dtype<<std::endl;
      std::cout<<"Dtype in src "<<l_join[k]->dtype<<std::endl;
      hipMemcpy(temp.data(), result_joincol[k]->data, sizeof(int64_t)*result_joincol[k]->size,hipMemcpyDeviceToHost);

      for (unsigned int i = 0; i < temp.size(); ++i)
      {
          std::cout<<i<<". "<<temp[i]<<std::endl;
      }
      }

    #endif
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(join_source_table,join_destination_table);
      CHECK_STREAM(0);
      CUDF_EXPECTS(update_err == GDF_SUCCESS, "nvcategory_gather_table throwing a GDF error");
    }

     
    std::cout<<"After join result"<<std::endl;
    
    POP_RANGE();
    return std::pair<cudf::table, cudf::table>(result_left, result_right);
}

template <JoinType join_type, typename index_type>
std::pair<cudf::table, cudf::table> join_call_compute_df(
                         cudf::table & left_cols, 
                         std::vector <int> left_join_cols,
                         cudf::table & right_cols,
                         std::vector <int> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <int> left_join_result_cols,
                         std::vector <int> right_join_result_cols) {

  int num_left_cols = left_cols.num_columns();
  int num_right_cols = right_cols.num_columns();
  int num_cols_to_join = left_join_cols.size();

  CUDF_EXPECTS (0 != num_left_cols, "Left table is empty");
  CUDF_EXPECTS (0 != num_right_cols, "Right table is empty");
  CUDF_EXPECTS (nullptr != join_context, "Join context is invalid");

  if (0 == num_cols_to_join)
  {
      std::cout<<"num of cols to join is 0"<<std::endl;
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {});
  }

  for(int column_index = 0; column_index  < num_left_cols; column_index++){
    CUDF_EXPECTS(left_cols.get_column(column_index)->dtype != GDF_invalid, "Unsupported Dtype in Left column");
  }
  for(int column_index = 0; column_index  < num_right_cols; column_index++){
    CUDF_EXPECTS(right_cols.get_column(column_index)->dtype != GDF_invalid, "Unsupported Dtype in right column");
  }

  auto const left_col_size = left_cols.get_column(0)->size;
  auto const right_col_size = right_cols.get_column(0)->size;

  // If the inputs are empty, immediately return
  if ((0 == left_col_size) && (0 == right_col_size)) {
      std::cout<<"Return from 2"<<std::endl;
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {});
  }

  // If left join and the left table is empty, return immediately
  if ((JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)) {
      std::cout<<"Return from 3"<<std::endl;
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {});
  }

  // If Inner Join and either table is empty, return immediately
  if ((JoinType::INNER_JOIN == join_type) &&
      ((0 == left_col_size) || (0 == right_col_size))) {
      std::cout<<"Return from 4"<<std::endl;
      return std::pair <cudf::table, cudf::table> (cudf::table {}, cudf::table {});
  }


  //if the inputs are nvcategory we need to make the dictionaries comparable
  bool at_least_one_category_column = false;
  for(int join_column_index = 0; join_column_index < num_cols_to_join; join_column_index++){
    at_least_one_category_column |= left_cols.get_column(left_join_cols[join_column_index])->dtype == GDF_STRING_CATEGORY;
  }

  std::vector<gdf_column*> new_left_cols(left_cols.begin(), left_cols.end());
  std::vector<gdf_column*> new_right_cols(right_cols.begin(), right_cols.end());
  std::vector<gdf_column *> temp_columns_to_free;
  if(at_least_one_category_column){
    for(int join_column_index = 0; join_column_index < num_cols_to_join; join_column_index++){
      if(left_cols.get_column(left_join_cols[join_column_index])->dtype == GDF_STRING_CATEGORY){
        CUDF_EXPECTS(right_cols.get_column(right_join_cols[join_column_index])->dtype == GDF_STRING_CATEGORY, "GDF type mismatch");

        gdf_column * left_original_column = new_left_cols[left_join_cols[join_column_index]];
        gdf_column * right_original_column = new_right_cols[right_join_cols[join_column_index]];




        gdf_column * new_left_column_ptr = new gdf_column{};
        gdf_column * new_right_column_ptr = new gdf_column{};

        temp_columns_to_free.push_back(new_left_column_ptr);
        temp_columns_to_free.push_back(new_right_column_ptr);


        gdf_column * input_join_columns_merge[2] = {left_original_column, right_original_column};
        gdf_column * new_join_columns[2] = {new_left_column_ptr,
            new_right_column_ptr};
        gdf_column_view(new_left_column_ptr, nullptr, nullptr, left_original_column->size, GDF_STRING_CATEGORY);
        gdf_column_view(new_right_column_ptr, nullptr, nullptr, right_original_column->size, GDF_STRING_CATEGORY);

        int col_width = cudf::byte_width(*new_left_column_ptr);
        RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->data), col_width * left_original_column->size, 0) ); // TODO: non-default stream?
        if(left_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(left_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_left_column_ptr->valid, left_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(left_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_left_column_ptr->valid = nullptr;
        }
        new_left_column_ptr->null_count = left_original_column->null_count;


        RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->data), col_width * right_original_column->size, 0) ); // TODO: non-default stream?
        if(right_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(right_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_right_column_ptr->valid, right_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(right_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_right_column_ptr->valid = nullptr;
        }
        new_right_column_ptr->null_count = right_original_column->null_count;
        gdf_error err = sync_column_categories(input_join_columns_merge,
            new_join_columns,
            2);

        CUDF_EXPECTS(GDF_SUCCESS == err, "GDF_ERROR");

        new_left_cols[left_join_cols[join_column_index]] = new_join_columns[0];
        new_right_cols[right_join_cols[join_column_index]] = new_join_columns[1];
        CHECK_STREAM(0);
      }
    }
  }

  cudf::table  updated_left_cols(new_left_cols);
  cudf::table  updated_right_cols(new_right_cols);
  

  // If index outputs are not requested, create columns to store them
  // for computing combined join output
  gdf_column *left_index_out = left_indices;
  gdf_column *right_index_out = right_indices;

  using gdf_col_pointer =
      typename std::unique_ptr<gdf_column, std::function<void(gdf_column *)>>;
  auto gdf_col_deleter = [](gdf_column *col) {
    col->size = 0;
    if (col->data) {
      RMM_FREE(col->data, 0);
    }
    if (col->valid) {
      RMM_FREE(col->valid, 0);
    }
  };
  gdf_col_pointer l_index_temp, r_index_temp;

  if (nullptr == left_indices) {
    l_index_temp = {new gdf_column{}, gdf_col_deleter};
    left_index_out = l_index_temp.get();
    }

    if (nullptr == right_indices) {
        r_index_temp = {new gdf_column{}, gdf_col_deleter};
        right_index_out = r_index_temp.get();
    }

    //get column pointers to join on
    std::vector<gdf_column*> ljoincol;
    std::vector<gdf_column*> rjoincol;
    for (int i = 0; i < num_cols_to_join; ++i) {
        ljoincol.push_back(updated_left_cols.get_column(left_join_cols[i]));
        rjoincol.push_back(updated_right_cols.get_column(right_join_cols[i]));
    }


    cudf::table ljoin_cols(ljoincol);
    cudf::table rjoin_cols(rjoincol);
    if (ljoin_cols.get_column(0) == nullptr)
        std::cout<<"RGSL : Null value"<<std::endl;
    join_call<join_type>(num_cols_to_join,
            ljoin_cols, rjoin_cols,
            left_index_out, right_index_out,
            join_context);
    CHECK_STREAM(0);

    std::pair<cudf::table, cudf::table> merged_result =
        construct_join_output_df<join_type, index_type>(
            ljoin_cols, rjoin_cols,
            updated_left_cols, left_join_cols,
            updated_right_cols, left_index_out, right_index_out, 
            left_join_result_cols, right_join_result_cols);
    CHECK_STREAM(0);
    l_index_temp.reset(nullptr);
    r_index_temp.reset(nullptr);





    //freeing up the temp column used to synch categories between columns
    for(unsigned int column_to_free = 0; column_to_free < temp_columns_to_free.size(); column_to_free++){
      gdf_column_free(temp_columns_to_free[column_to_free]);
      delete temp_columns_to_free[column_to_free];
    }

    CHECK_STREAM(0);
    
    std::cout<<"Returning merged result"<<std::endl;
    return merged_result;
}

std::pair<cudf::table, cudf::table> gdf_left_join(
                         cudf::table & left_cols,
                         std::vector <int> left_join_cols,
                         cudf::table & right_cols,
                         std::vector <int> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <int> left_join_result_cols,
                         std::vector <int> right_join_result_cols) {
    return join_call_compute_df<JoinType::LEFT_JOIN, output_index_type>(
                     left_cols, 
                     left_join_cols,
                     right_cols,
                     right_join_cols,
                     left_indices,
                     right_indices,
                     join_context,
                     left_join_result_cols,
                     right_join_result_cols);
}

std::pair<cudf::table, cudf::table> gdf_inner_join(
                         cudf::table & left_cols,
                         std::vector <int> left_join_cols,
                         cudf::table & right_cols,
                         std::vector <int> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <int> left_join_result_cols,
                         std::vector <int> right_join_result_cols) {
    return join_call_compute_df<JoinType::INNER_JOIN, output_index_type>(
                     left_cols,
                     left_join_cols,
                     right_cols,
                     right_join_cols,
                     left_indices,
                     right_indices,
                     join_context,
                     left_join_result_cols,
                     right_join_result_cols);
}

std::pair<cudf::table, cudf::table> gdf_full_join(
                         cudf::table & left_cols,
                         std::vector <int> left_join_cols,
                         cudf::table & right_cols,
                         std::vector <int> right_join_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context,
                         std::vector <int> left_join_result_cols,
                         std::vector <int> right_join_result_cols) {
    return join_call_compute_df<JoinType::FULL_JOIN, output_index_type>(
                     left_cols,
                     left_join_cols,
                     right_cols,
                     right_join_cols,
                     left_indices,
                     right_indices,
                     join_context,
                     left_join_result_cols,
                     right_join_result_cols);
}
}
